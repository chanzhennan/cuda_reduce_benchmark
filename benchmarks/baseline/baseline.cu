#include "hip/hip_runtime.h"
#include "baseline/baseline.cuh"

template <size_t blockSize, typename T>
__global__ void reducebase(T *g_idata, T *g_odata, size_t size)
{
  __shared__ T sdata[blockSize];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = 0;
  if(i<size)
    sdata[tid] = g_idata[i];
  __syncthreads();

  for(unsigned int s=1; s < blockDim.x; s *= 2) {
      if (tid % (2*s) == 0) {
        sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
    }

   if (tid == 0) g_odata[blockIdx.x] = sdata[0];


}

// PRE:
// dA is an array allocated on the GPU
// N <= len(dA) is a power of two (N >= BLOCKSIZE)
// POST: the sum of the first N elements of dA is returned
template<size_t blockSize, typename T>
T GPUReduction(T* dA, size_t N)
{
   int size = N;
   // thrust::host_vector<int> data_h_i(size, 1);

   int threadsPerBlock = 256;
   int totalBlocks = (size + (threadsPerBlock - 1)) / threadsPerBlock;

   T *output;
   hipMalloc((void **)&output, sizeof(T) * totalBlocks);

   bool turn = true;

   while (true)
   {
      if (turn)
      {
         reducebase<blockSize><<<totalBlocks, threadsPerBlock>>>(dA, output, size);
   hipDeviceSynchronize();

         turn = false;
       }
       else{
         reducebase<blockSize><<<totalBlocks, threadsPerBlock>>>(output, dA, size);
   hipDeviceSynchronize();

         turn = true;
       }

       if(totalBlocks == 1) break;
       size = totalBlocks;
       totalBlocks = ceil((double)totalBlocks/threadsPerBlock);

     }

     T tot = 0.;

     if(turn)
     {
       hipMemcpy(&tot, dA, sizeof(T), hipMemcpyDeviceToHost);
     }
     else
     {
       hipMemcpy(&tot, output, sizeof(T), hipMemcpyDeviceToHost);
     }
     hipFree(output);
     //  std::cout << tot << std::endl;

     return tot;


}


template float GPUReduction<1024, float>(float *dA, size_t N);
template int GPUReduction<1024, int>(int *dA, size_t N);
