#include "hip/hip_runtime.h"
#include "ReduceArray2/no_bank_conflict.cuh"

template <size_t blockSize, typename T>
__global__ void reducebase3(T *g_idata, T *g_odata, size_t size) {
  __shared__ T sdata[blockSize];

  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = 0;

  if (i < size) sdata[tid] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// PRE:
// dA is an array allocated on the GPU
// N <= len(dA) is a power of two (N >= BLOCKSIZE)
// POST: the sum of the first N elements of dA is returned
template <size_t blockSize, typename T>
T GPUReduction3(T *dA, size_t N) {
  int size = N;
  // thrust::host_vector<int> data_h_i(size, 1);

  int totalBlocks = (size + (TPB - 1)) / TPB;

  T *output;
  hipMalloc((void **)&output, sizeof(T) * totalBlocks);

  bool turn = true;

  while (true) {
    if (turn) {
      reducebase3<blockSize><<<totalBlocks, TPB>>>(dA, output, size);
      turn = false;
    } else {
      reducebase3<blockSize><<<totalBlocks, TPB>>>(output, dA, size);
      turn = true;
    }

    if (totalBlocks == 1) break;
    size = totalBlocks;
    totalBlocks = ceil((double)totalBlocks / TPB);
  }
  hipDeviceSynchronize();

  T tot = 0.;

  if (turn) {
    hipMemcpy(&tot, dA, sizeof(T), hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(&tot, output, sizeof(T), hipMemcpyDeviceToHost);
  }
  hipFree(output);
  //  std::cout << tot << std::endl;

  return tot;
}

template float GPUReduction3<TPB, float>(float *dA, size_t N);
template int GPUReduction3<TPB, int>(int *dA, size_t N);
